#include "hip/hip_runtime.h"
#include "../student/utils.h"

#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
  
__global__
void simpleHisto(const unsigned int* const vals,
                 unsigned int* const histo,
                 int numVals)
{
  const int blockId = (blockIdx.y * gridDim.x + blockIdx.x);
  const int tid = blockId * blockDim.x + threadIdx.x;
  if (tid >= numVals)
    return;

  unsigned int bin = vals[tid];

  atomicAdd(histo + bin, 1);
}

//we launch a small number of blocks that
//go through the entire memory accumulating
//results in the shared memory
template<int numBlocks, int numThreads>
__global__
void fasterHisto1(const unsigned int* const vals,
                  unsigned int* const histo,
                  const unsigned int numVals,
                  const unsigned int numBins)
{
  extern __shared__ int s_bins[];

  const int tid = threadIdx.x + blockIdx.x * numThreads;

  //zero out smem
  #pragma unroll
  for (int i = threadIdx.x ; i < numBins; i += numThreads) {
    s_bins[i] = 0;
  }

  __syncthreads();

  //loop through vals and accum to shared memory
  #pragma unroll
  for (unsigned int i = tid; i < numVals; i += numBlocks * numThreads) {
    unsigned int bin = vals[i];

    atomicAdd(&s_bins[bin], 1);
  }

  __syncthreads();

  //atomically update global histo
  #pragma unroll
  for (int i = threadIdx.x; i < numBins; i += numThreads) {
    atomicAdd(histo + i, s_bins[i]);
  }
}

void computeHistogram(const unsigned int* const d_vals,
                      unsigned int* const d_histo,
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  const unsigned int numThreads = 192;
  const unsigned int numBlocks = 80;

  //grid needs to be 2D to handle large number of elements

  int side = ceil(sqrt(numElems / (double)numThreads));
  dim3 gridSize(side, side, 1);
  //call kernel
  /////////////////////////////////////////////
  //Solution 1, basic global atomic increment
  simpleHisto<<< gridSize, numThreads>>>(d_vals, d_histo, numElems);

  ///////////////////////////////////////////////////////////
  //Solution 2, using shared mem atomics
  //fasterHisto1<numBlocks, numThreads><<<numBlocks, numThreads, numBins * sizeof(unsigned int)>>>(d_vals, d_histo, numElems, numBins);

  ////////////////////////////////////////////////////////////
  //Solution 3, with thrust and sorting
  //Theoretically doing a full sort for a histogram is overkill
  /*thrust::device_ptr<unsigned int> dv((unsigned int *)d_vals);

  thrust::sort(dv, dv + numElems);

  thrust::upper_bound(dv, dv + numElems,
                      thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator(numBins),
                      thrust::device_ptr<unsigned int>(d_histo));

  thrust::adjacent_difference(thrust::device_ptr<unsigned int>(d_histo), thrust::device_ptr<unsigned int>(d_histo) + numBins,
                              thrust::device_ptr<unsigned int>(d_histo)); */

  /////////////////////////////////////////////////////////////
  //Solution 4
  //Use a 1/100 sampling to determine mean, then use registers
  //for accumulation of values around mean to reduce contention
  //To be implemented

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
